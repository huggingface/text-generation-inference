#include "hip/hip_runtime.h"
#include "q_matrix.cuh"
#include "matrix_view.cuh"
#include "util.cuh"

#include "quant/qdq_2.cuh"
#include "quant/qdq_3.cuh"
#include "quant/qdq_4.cuh"
#include "quant/qdq_5.cuh"
#include "quant/qdq_6.cuh"
#include "quant/qdq_8.cuh"

#define BLOCK_KN_SIZE 128

#define THREADS_X 32
#define THREADS_Y 32

// Shuffle quantized data on load

__global__ void shuffle_kernel
(
    uint32_t* __restrict__ b_q_weight,
    const int size_k,
    const int size_n,
    const int rows_8,
    const int rows_6,
    const int rows_5,
    const int rows_4,
    const int rows_3,
    const int rows_2
)
{
    int n = blockIdx.x * THREADS_X + threadIdx.x;
    if (n >= size_n) return;
    int k = 0;
    uint32_t* b_ptr = b_q_weight + n;
    while (k < rows_8) { shuffle_8bit_4 (b_ptr, size_n); b_ptr += 1 * size_n; k +=  4; }
    while (k < rows_6) { shuffle_6bit_16(b_ptr, size_n); b_ptr += 3 * size_n; k += 16; }
    while (k < rows_5) { shuffle_5bit_32(b_ptr, size_n); b_ptr += 5 * size_n; k += 32; }
    while (k < rows_4) { shuffle_4bit_8 (b_ptr, size_n); b_ptr += 1 * size_n; k +=  8; }
    while (k < rows_3) { shuffle_3bit_32(b_ptr, size_n); b_ptr += 3 * size_n; k += 32; }
    while (k < rows_2) { shuffle_2bit_16(b_ptr, size_n); b_ptr += 1 * size_n; k += 16; }
}


// QMatrix constructor

QMatrix::QMatrix
(
    const int _device,
    const int _height,
    const int _width,
    const int _groups,

    uint32_t* _q_weight,
    uint16_t* _q_perm,
    uint16_t* _q_invperm,
    uint32_t* _q_scale,
    half* _q_scale_max,
    uint16_t* _q_groups,
    uint16_t* _q_group_map,

    uint32_t* _gptq_qzeros,
    half* _gptq_scales,
    uint32_t* _gptq_g_idx,

    half* _temp_dq
) :
    device(_device),
    height(_height),
    width(_width),
    groups(_groups),
    temp_dq(_temp_dq)
{
    hipSetDevice(device);

    failed = false;

    cuda_q_weight = _q_weight;
    cuda_q_perm = _q_perm;
    cuda_q_invperm = _q_invperm;
    cuda_q_scale = _q_scale;
    cuda_q_scale_max = _q_scale_max;
    cuda_q_groups = _q_groups;
    cuda_q_group_map = _q_group_map;
    cuda_gptq_qzeros = _gptq_qzeros;
    cuda_gptq_scales = _gptq_scales;

    is_gptq = (_gptq_qzeros != NULL);

    if (is_gptq)
    {
        gptq_groupsize = 1;
        while (gptq_groupsize * groups < height) gptq_groupsize *= 2;
    }

    // Create group map

    rows_8 = 0;
    rows_6 = 0;
    rows_5 = 0;
    rows_4 = 0;
    rows_3 = 0;
    rows_2 = 0;

    if (!is_gptq)
    {
        uint16_t* cpu_q_groups = (uint16_t*)calloc(groups * 2, sizeof(uint16_t));
        hipMemcpy(cpu_q_groups, cuda_q_groups, groups * 2 * sizeof(uint16_t), hipMemcpyDeviceToHost);

        int row = 0;
        for (int i = 0; i < groups; i++)
        {
            int bits = cpu_q_groups[i * 2];

            int rows;
            if (i < groups - 1)
            {
                int qrows = cpu_q_groups[i * 2 + 3] - cpu_q_groups[i * 2 + 1];
                rows = qrows * 32 / bits;
            }
            else rows = height - row;

            if (bits == 8) rows_8 += rows;
            if (bits == 6) rows_6 += rows;
            if (bits == 5) rows_5 += rows;
            if (bits == 4) rows_4 += rows;
            if (bits == 3) rows_3 += rows;
            if (bits == 2) rows_2 += rows;
            row += rows;
        }

        free(cpu_q_groups);

        rows_6 += rows_8;
        rows_5 += rows_6;
        rows_4 += rows_5;
        rows_3 += rows_4;
        rows_2 += rows_3;
    }
    else
    {
        rows_4 = height;
        rows_3 = height;
        rows_2 = height;

        if (_gptq_g_idx)
        {
            if (!make_sequential(_gptq_g_idx))
            {
                failed = true;
                //printf("FAIL\n");
                return;
            }
        }
    }

//     DBGI(rows_8);
//     DBGI(rows_6);
//     DBGI(rows_5);
//     DBGI(rows_4);
//     DBGI(rows_3);
//     DBGI(rows_2);

    // Shuffle quantized data

    dim3 blockDim, gridDim;
    blockDim.x = THREADS_X;
    blockDim.y = 1;
    gridDim.x = DIVIDE(width, THREADS_X);
    gridDim.y = 1;
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();

    shuffle_kernel<<<gridDim, blockDim, 0, stream>>>(cuda_q_weight, height, width, rows_8, rows_6, rows_5, rows_4, rows_3, rows_2);
}

QMatrix::~QMatrix()
{
}

// Reconstruct b[k,n] (GPTQ)

__global__ void reconstruct_gptq_kernel
(
    const uint32_t* __restrict__ b_q_weight,
    const uint16_t* __restrict__ b_q_perm,
    const uint32_t* __restrict__ b_gptq_qzeros,
    const half* __restrict__ b_gptq_scales,
    //const uint16_t* __restrict__ b_q_groups,
    const int size_k,
    const int size_n,
    const int groupsize,
    const int groups,
    half* __restrict__ b,
    const int rows_4
)
{
    MatrixView_half_rw b_(b, size_k, size_n);
    MatrixView_q4_row b_gptq_qzeros_(b_gptq_qzeros, groups, size_n);
    MatrixView_half b_gptq_scales_(b_gptq_scales, groups, size_n);

    int offset_k = BLOCK_KN_SIZE * blockIdx.y;
    int offset_n = BLOCK_KN_SIZE * blockIdx.x * 4;

    int end_k = min(offset_k + BLOCK_KN_SIZE, size_k);

    // Preload remapping table

    __shared__ uint16_t perm[BLOCK_KN_SIZE];
    int t = threadIdx.x;

    if (b_q_perm)
    {
        if (offset_k + t < size_k)
            perm[t] = b_q_perm[offset_k + t];
    }

    // Column

    int n = offset_n + t * 4;
    if (n >= size_n) return;

    // Find initial group

    int group = offset_k / groupsize;
    int nextgroup = offset_k + groupsize;

    // b offset

    int qk = offset_k / (32 / 4);

    const uint32_t* b_ptr = b_q_weight + qk * size_n + n;

    // Initial zeros/scale

    int zeros[4];
    half2 scales[4];
    half2 z1z16[4][2];
    half2 y1y16[4][2];
    b_gptq_qzeros_.item4(zeros, group, n);
    b_gptq_scales_.item4_h2(scales, group, n);
    dequant_4bit_8_prep_zero((zeros[0] + 1) & 0x0F, z1z16[0], y1y16[0]);
    dequant_4bit_8_prep_zero((zeros[1] + 1) & 0x0F, z1z16[1], y1y16[1]);
    dequant_4bit_8_prep_zero((zeros[2] + 1) & 0x0F, z1z16[2], y1y16[2]);
    dequant_4bit_8_prep_zero((zeros[3] + 1) & 0x0F, z1z16[3], y1y16[3]);

    __syncthreads();

    int k = offset_k;
    int lk = 0;

    while (k < end_k)
    {
        if (k == nextgroup)
        {
            group++;
            nextgroup += groupsize;
            b_gptq_qzeros_.item4(zeros, group, n);
            b_gptq_scales_.item4_h2(scales, group, n);
            dequant_4bit_8_prep_zero((zeros[0] + 1) & 0x0F, z1z16[0], y1y16[0]);
            dequant_4bit_8_prep_zero((zeros[1] + 1) & 0x0F, z1z16[1], y1y16[1]);
            dequant_4bit_8_prep_zero((zeros[2] + 1) & 0x0F, z1z16[2], y1y16[2]);
            dequant_4bit_8_prep_zero((zeros[3] + 1) & 0x0F, z1z16[3], y1y16[3]);
        }

        for (int p = 0; p < 4; p++)
        {
            half2 dq[4][4];
            const int4* b_ptr4 = (int4*) b_ptr;
            int4 load_int4 = *b_ptr4;

            dequant_4bit_8_gptq(load_int4.x, dq[0], z1z16[0], y1y16[0], size_n, false);
            dequant_4bit_8_gptq(load_int4.y, dq[1], z1z16[1], y1y16[1], size_n, false);
            dequant_4bit_8_gptq(load_int4.z, dq[2], z1z16[2], y1y16[2], size_n, false);
            dequant_4bit_8_gptq(load_int4.w, dq[3], z1z16[3], y1y16[3], size_n, false);

            b_ptr += size_n;
            //half* dqh = (half*)dq;
            if (b_q_perm)
            {
                for (int j = 0; j < 4; j++)
                {
                    for (int v = 0; v < 4; v++) dq[v][j] = __hmul2(scales[v], dq[v][j]);
                    b_.set4(perm[lk++], n, __low2half(dq[0][j]), __low2half(dq[1][j]), __low2half(dq[2][j]), __low2half(dq[3][j]));
                    b_.set4(perm[lk++], n, __high2half(dq[0][j]), __high2half(dq[1][j]), __high2half(dq[2][j]), __high2half(dq[3][j]));
                }
            }
            else
            {
                for (int j = 0; j < 4; j++)
                {
                    for (int v = 0; v < 4; v++) dq[v][j] = __hmul2(scales[v], dq[v][j]);
                    b_.set4(offset_k + lk++, n, __low2half(dq[0][j]), __low2half(dq[1][j]), __low2half(dq[2][j]), __low2half(dq[3][j]));
                    b_.set4(offset_k + lk++, n, __high2half(dq[0][j]), __high2half(dq[1][j]), __high2half(dq[2][j]), __high2half(dq[3][j]));
                }
            }
        }
        k += 32;
    }
}


// Reconstruct b[k,n]

__global__ void reconstruct_kernel
(
    const uint32_t* __restrict__ b_q_weight,
    const uint16_t* __restrict__ b_q_perm,
    const uint32_t* __restrict__ b_q_scale,
    const half* __restrict__ b_q_scale_max,
    const uint16_t* __restrict__ b_q_group_map,
    const int size_k,
    const int size_n,
    //const int groupsize,
    const int groups,
    half* __restrict__ b,
    const int rows_8,
    const int rows_6,
    const int rows_5,
    const int rows_4,
    const int rows_3,
    const int rows_2
)
{
    MatrixView_half_rw b_(b, size_k, size_n);
    MatrixView_q4_row b_q_scale_(b_q_scale, groups, size_n);

    int offset_k = BLOCK_KN_SIZE * blockIdx.y;
    int offset_n = BLOCK_KN_SIZE * blockIdx.x;

    // Preload remapping table

    int t = threadIdx.x;
    __shared__ uint16_t perm[BLOCK_KN_SIZE];
    if (offset_k + t < size_k)
        perm[t] = b_q_perm[offset_k + t];

    // Column

    int n = offset_n + t;
    if (n >= size_n) return;

    // Find initial group

    // int group = offset_k / groupsize;
    int group = b_q_group_map[offset_k * 2];

    int pre_rows_8 = min(rows_8, offset_k);
    int pre_rows_6 = offset_k > rows_8 ? min(rows_6, offset_k) - rows_8 : 0;
    int pre_rows_5 = offset_k > rows_6 ? min(rows_5, offset_k) - rows_6 : 0;
    int pre_rows_4 = offset_k > rows_5 ? min(rows_4, offset_k) - rows_5 : 0;
    int pre_rows_3 = offset_k > rows_4 ? min(rows_3, offset_k) - rows_4 : 0;
    int pre_rows_2 = offset_k > rows_3 ? min(rows_2, offset_k) - rows_3 : 0;
    int qk = 0;
    qk += pre_rows_8 / 32 * 8;
    qk += pre_rows_6 / 32 * 6;
    qk += pre_rows_5 / 32 * 5;
    qk += pre_rows_4 / 32 * 4;
    qk += pre_rows_3 / 32 * 3;
    qk += pre_rows_2 / 32 * 2;

    const uint32_t* b_ptr = b_q_weight + qk * size_n + n;

    half qs_h = dq_scale(b_q_scale_.item(group, n), b_q_scale_max[group]);
    half2 qs_h2 = __halves2half2(qs_h, qs_h);
    int nextgroup = offset_k + b_q_group_map[offset_k * 2 + 1];

    int end_k = min(offset_k + BLOCK_KN_SIZE, size_k);
    int k = offset_k;
    int lk = 0;

    __syncthreads();

    while (k < rows_8 && k < end_k)
    {
        if (k == nextgroup) { group++; qs_h = dq_scale(b_q_scale_.item(group, n), b_q_scale_max[group]); nextgroup += b_q_group_map[k * 2 + 1]; qs_h2 = __halves2half2(qs_h, qs_h); }
        for (int p = 0; p < 4; p++)
        {
            half2 dq[4];
            uint32_t q_0 = *b_ptr; b_ptr += size_n;
            uint32_t q_1 = *b_ptr; b_ptr += size_n;
            dequant_8bit_8(q_0, q_1, dq, size_n);
            for (int j = 0; j < 4; j++) dq[j] = __hmul2(dq[j], qs_h2);
            half* dqh = (half*) dq;
            for (int j = 0; j < 8; j++) b_.set(perm[lk++], n, dqh[j]);
        }
        k += 32;
    }

    while (k < rows_6 && k < end_k)
    {
        if (k == nextgroup) { group++; qs_h = dq_scale(b_q_scale_.item(group, n), b_q_scale_max[group]); nextgroup += b_q_group_map[k * 2 + 1]; qs_h2 = __halves2half2(qs_h, qs_h); }
        for (int p = 0; p < 2; p++)
        {
            half2 dq[8];
            uint32_t q_0 = *b_ptr; b_ptr += size_n;
            uint32_t q_1 = *b_ptr; b_ptr += size_n;
            uint32_t q_2 = *b_ptr; b_ptr += size_n;
            dequant_6bit_16(q_0, q_1, q_2, dq, size_n);
            for (int j = 0; j < 8; j++) dq[j] = __hmul2(dq[j], qs_h2);
            half* dqh = (half*) dq;
            for (int j = 0; j < 16; j++) b_.set(perm[lk++], n, dqh[j]);
        }
        k += 32;
    }

    while (k < rows_5 && k < end_k)
    {
        if (k == nextgroup) { group++; qs_h = dq_scale(b_q_scale_.item(group, n), b_q_scale_max[group]); nextgroup += b_q_group_map[k * 2 + 1]; qs_h2 = __halves2half2(qs_h, qs_h); }
        for (int p = 0; p < 1; p++)
        {
            half2 dq[16];
            uint32_t q_0 = *b_ptr; b_ptr += size_n;
            uint32_t q_1 = *b_ptr; b_ptr += size_n;
            uint32_t q_2 = *b_ptr; b_ptr += size_n;
            uint32_t q_3 = *b_ptr; b_ptr += size_n;
            uint32_t q_4 = *b_ptr; b_ptr += size_n;
            dequant_5bit_32(q_0, q_1, q_2, q_3, q_4, dq, size_n);
            for (int j = 0; j < 16; j++) dq[j] = __hmul2(dq[j], qs_h2);
            half* dqh = (half*) dq;
            for (int j = 0; j < 32; j++) b_.set(perm[lk++], n, dqh[j]);
        }
        k += 32;
    }

    while (k < rows_4 && k < end_k)
    {
        if (k == nextgroup) { group++; qs_h = dq_scale(b_q_scale_.item(group, n), b_q_scale_max[group]); nextgroup += b_q_group_map[k * 2 + 1]; qs_h2 = __halves2half2(qs_h, qs_h); }
        for (int p = 0; p < 4; p++)
        {
            half2 dq[4];
            uint32_t q_0 = *b_ptr; b_ptr += size_n;
            dequant_4bit_8(q_0, dq, size_n);
            for (int j = 0; j < 4; j++) dq[j] = __hmul2(dq[j], qs_h2);
            half* dqh = (half*) dq;
            for (int j = 0; j < 8; j++) b_.set(perm[lk++], n, dqh[j]);
        }
        k += 32;
    }

    while (k < rows_3 && k < end_k)
    {
        if (k == nextgroup) { group++; qs_h = dq_scale(b_q_scale_.item(group, n), b_q_scale_max[group]); nextgroup += b_q_group_map[k * 2 + 1]; qs_h2 = __halves2half2(qs_h, qs_h); }
        for (int p = 0; p < 1; p++)
        {
            half2 dq[16];
            uint32_t q_0 = *b_ptr; b_ptr += size_n;
            uint32_t q_1 = *b_ptr; b_ptr += size_n;
            uint32_t q_2 = *b_ptr; b_ptr += size_n;
            dequant_3bit_32(q_0, q_1, q_2, dq, size_n);
            for (int j = 0; j < 16; j++) dq[j] = __hmul2(dq[j], qs_h2);
            half* dqh = (half*) dq;
            for (int j = 0; j < 32; j++) b_.set(perm[lk++], n, dqh[j]);
        }
        k += 32;
    }

    while (k < rows_2 && k < end_k)
    {
        if (k == nextgroup) { group++; qs_h = dq_scale(b_q_scale_.item(group, n), b_q_scale_max[group]); nextgroup += b_q_group_map[k * 2 + 1]; qs_h2 = __halves2half2(qs_h, qs_h); }
        for (int p = 0; p < 1; p++)
        {
            half2 dq[8];
            uint32_t q_0 = *b_ptr; b_ptr += size_n;
            dequant_2bit_16(q_0, dq, size_n);
            for (int j = 0; j < 8; j++) dq[j] = __hmul2(dq[j], qs_h2);
            half* dqh = (half*) dq;
            for (int j = 0; j < 16; j++) b_.set(perm[lk++], n, dqh[j]);
        }
        k += 16;
    }
}

void QMatrix::reconstruct(half* out)
{
    dim3 blockDim, gridDim;
    blockDim.x = BLOCK_KN_SIZE;
    blockDim.y = 1;
    gridDim.y = DIVIDE(height, BLOCK_KN_SIZE);
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();

    if (!is_gptq)
    {
        gridDim.x = DIVIDE(width, BLOCK_KN_SIZE);
        reconstruct_kernel<<<gridDim, blockDim, 0, stream>>>
        (
            cuda_q_weight,
            cuda_q_perm,
            cuda_q_scale,
            cuda_q_scale_max,
            cuda_q_group_map,
            height,
            width,
            //groupsize,
            groups,
            out,
            rows_8,
            rows_6,
            rows_5,
            rows_4,
            rows_3,
            rows_2
        );
    }
    else
    {
        gridDim.x = DIVIDE(width, BLOCK_KN_SIZE * 4);
        reconstruct_gptq_kernel<<<gridDim, blockDim, 0, stream>>>
        (
            cuda_q_weight,
            cuda_q_perm,
            cuda_gptq_qzeros,
            cuda_gptq_scales,
            //const uint16_t* __restrict__ b_q_groups,
            height,
            width,
            gptq_groupsize,
            groups,
            out,
            rows_4
        );
    }
}

__global__ void make_sequential_kernel
(
    const uint32_t* __restrict__ w,
    uint32_t* __restrict__ w_new,
    const uint16_t* __restrict__ q_perm,
    const int w_height,
    const int w_width
)
{
    const uint64_t* w2 = (uint64_t*) w;
    uint64_t* w_new2 = (uint64_t*) w_new;
    int w2_stride = w_width >> 1;

    int w2_column = THREADS_X * blockIdx.x + threadIdx.x;
    if (w2_column >= w2_stride) return;

    int w_new2_row = blockIdx.y;

    int q_perm_idx = w_new2_row << 3;

    uint64_t dst = 0;

    #pragma unroll
    for (int i = 0; i < 8; i++)
    {
        int source_row = q_perm[q_perm_idx++];

        int w2_row = source_row >> 3;
        int w2_subrow = source_row & 0x07;
        int w2_row_shift = w2_subrow << 2;
        int wnew2_row_shift = i << 2;

        uint64_t src = w2[w2_row * w2_stride + w2_column];
        src >>= w2_row_shift;
        src &= 0x0000000f0000000f;
        src <<= wnew2_row_shift;
        dst |= src;
    }

    w_new2[w_new2_row * w2_stride + w2_column] = dst;
}

bool QMatrix::make_sequential(const uint32_t* cpu_g_idx)
{
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();
    uint32_t* cuda_new_qweight = NULL;
    hipError_t err = hipMalloc(&cuda_new_qweight, height / 8 * width * sizeof(uint32_t));
    if (err != hipSuccess) {
        hipError_t cuda_status = hipGetLastError(); // Clear error
        return false;
    }

    uint32_t* cpu_g_idx_map = (uint32_t*) calloc(groups, sizeof(uint32_t));
    uint32_t* cpu_x_map = (uint32_t*) malloc(height * sizeof(uint32_t));
    uint32_t* cpu_x_map_inv = (uint32_t*) malloc(height * sizeof(uint32_t));

    // Group histogram

    for (int i = 0; i < height; i++) cpu_g_idx_map[cpu_g_idx[i]]++;

    // Group map

    for (int i = 0, acc = 0; i < groups; i++)
    {
        short tmp = cpu_g_idx_map[i];
        cpu_g_idx_map[i] = acc;
        acc += tmp;
    }

    // X map (inverse)

    for (int row = 0; row < height; row++)
    {
        uint32_t target_group = cpu_g_idx[row];
        uint32_t target_row = cpu_g_idx_map[target_group];
        cpu_g_idx_map[target_group]++;
        cpu_x_map_inv[row] = target_row;
    }

    // X map

    for (int row = 0; row < height; row++) cpu_x_map[cpu_x_map_inv[row]] = row;

    // Reduce to uint16_t

    uint16_t* cpu_x_map16 = (uint16_t*)cpu_x_map;
    uint16_t* cpu_x_map_inv16 = (uint16_t*)cpu_x_map_inv;
    for (int row = 0; row < height; row++) cpu_x_map16[row] = (uint16_t) cpu_x_map[row];
    for (int row = 0; row < height; row++) cpu_x_map_inv16[row] = (uint16_t) cpu_x_map_inv[row];

    // Move to CUDA

    hipMemcpyAsync(cuda_q_perm, cpu_x_map16, height * sizeof(uint16_t), hipMemcpyHostToDevice);
    hipMemcpyAsync(cuda_q_invperm, cpu_x_map_inv16, height * sizeof(uint16_t), hipMemcpyHostToDevice);

    // Rearrange rows in w

    dim3 blockDim, gridDim;
    blockDim.x = THREADS_X;
    blockDim.y = 1;
    gridDim.x = DIVIDE(width, THREADS_X);
    gridDim.y = height / 8;

    make_sequential_kernel<<<gridDim, blockDim, 0, stream>>>
    (
        cuda_q_weight,
        cuda_new_qweight,
        cuda_q_perm,
        height / 8,
        width
    );

    // Replace qweights

    hipMemcpyAsync(cuda_q_weight, cuda_new_qweight, height / 8 * width * sizeof(uint32_t), hipMemcpyDeviceToDevice);

    // Cleanup

    hipDeviceSynchronize();

    hipFree(cuda_new_qweight);
    free(cpu_g_idx_map);
    free(cpu_x_map);
    free(cpu_x_map_inv);

    return true;
}
