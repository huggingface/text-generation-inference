#include "hip/hip_runtime.h"
#include "q_gemm.cuh"
#include "util.cuh"
#include "matrix_view.cuh"
#include "../config.h"

#include "quant/qdq_2.cuh"
#include "quant/qdq_3.cuh"
#include "quant/qdq_4.cuh"
#include "quant/qdq_5.cuh"
#include "quant/qdq_6.cuh"
#include "quant/qdq_8.cuh"

#define GPTQ_BLOCK_KN_SIZE 128
#define GPTQ_BLOCK_M_SIZE_MAX 8
#define GPTQ_MAX_GROUPS_IN_BLOCK (GPTQ_BLOCK_KN_SIZE / 32)

#define EXL2_BLOCK_KN_SIZE 64
#define EXL2_BLOCK_M_SIZE_MAX 8
#define EXL2_MAX_GROUPS_IN_BLOCK (EXL2_BLOCK_KN_SIZE / 32)

#define CLEAR_N_SIZE 256

#include "q_gemm_kernel.cuh"
#include "q_gemm_kernel_gptq.cuh"

void gemm_half_q_half_cuda_part
(
    const half* a,
    QMatrix* b,
    half* c,
    int size_m,
    int size_n,
    int size_k,
    int m_count,
    bool clear,
    const half* r_weights,
    int r_weights_stride,
    bool mul_r_weights
)
{
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();
    if (!b->is_gptq)
    {
        dim3 blockDim, gridDim;
        blockDim.x = EXL2_BLOCK_KN_SIZE;
        blockDim.y = 1;
        blockDim.z = 1;
        gridDim.x = DIVIDE(size_n, EXL2_BLOCK_KN_SIZE * 4);
        gridDim.y = DIVIDE(size_m, m_count);
        gridDim.z = DIVIDE(size_k, EXL2_BLOCK_KN_SIZE);

        fp_gemm_half_q_half_kernel kernel = pick_gemm_half_q_half_kernel(m_count, r_weights != NULL, mul_r_weights);

        kernel<<<gridDim, blockDim, 0, stream>>>
        (
            a,
            b->cuda_q_weight,
            b->cuda_q_scale,
            b->cuda_q_scale_max,
            c,
            size_m,
            size_n,
            size_k,
            b->groups,
            b->cuda_q_group_map,
            b->cuda_q_perm,
            b->rows_8,
            b->rows_6,
            b->rows_5,
            b->rows_4,
            b->rows_3,
            b->rows_2,
            clear,
            r_weights,
            r_weights_stride
        );
    }
    else
    {
        dim3 blockDim, gridDim;
        blockDim.x = GPTQ_BLOCK_KN_SIZE;
        blockDim.y = 1;
        blockDim.z = 1;
        gridDim.x = DIVIDE(size_n, GPTQ_BLOCK_KN_SIZE * 4);
        gridDim.y = DIVIDE(size_m, m_count);
        gridDim.z = DIVIDE(size_k, GPTQ_BLOCK_KN_SIZE);

        fp_gemm_half_q_half_gptq_kernel kernel = pick_gemm_half_q_half_gptq_kernel(m_count, r_weights != NULL, mul_r_weights);

//         DBGX((uint64_t) r_weights);
//         if (r_weights)
//             print_global_mem(r_weights, 1, 1, 1);
//         DBGI(r_weights_stride);

        kernel<<<gridDim, blockDim, 0, stream>>>
        (
            a,
            b->cuda_q_weight,
            b->cuda_gptq_qzeros,
            b->cuda_gptq_scales,
            c,
            size_m,
            size_n,
            size_k,
            b->groups,
            b->gptq_groupsize,
            b->cuda_q_perm,
            b->rows_4,
            clear,
            r_weights,
            r_weights_stride
        );
    }
}

void gemm_half_q_half_cuda
(
    hipblasHandle_t cublas_handle,
    const half* a,
    QMatrix* b,
    half* c,
    int size_m,
    int size_n,
    int size_k,
    bool clear,
    half* temp_dq,
    bool force_cuda,
    const half* r_weights,
    const int r_weights_stride,
    bool mul_r_weights
)
{
    if (size_m > MAX_Q_GEMM_ROWS && !force_cuda)
    {
        // Reconstruct FP16 matrix, then cuBLAS

        if (!temp_dq) temp_dq = b->temp_dq;
        b->reconstruct(temp_dq);

        //hipblasSetMathMode(cublas_handle, HIPBLAS_TENSOR_OP_MATH);

        const half alpha = __float2half(1.0f);
        const half beta = clear ? __float2half(0.0f) : __float2half(1.0f);
        hipblasHgemm(cublas_handle,
                    HIPBLAS_OP_N,
                    HIPBLAS_OP_N,
                    size_n, size_m, size_k,
                    &alpha, temp_dq, size_n,
                            a,       size_k,
                    &beta,  c,       size_n);

        //const float alpha = 1.0f;
        //const float beta = clear ? 0.0f : 1.0f;
        //cublasSgemmEx(cublas_handle,
        //             HIPBLAS_OP_N,
        //             HIPBLAS_OP_N,
        //             size_n, size_m, size_k,
        //             &alpha, temp_dq, HIP_R_16F, size_n,
        //                     a,       HIP_R_16F, size_k,
        //             &beta,  c,       HIP_R_16F, size_n);

        //const float alpha = 1.0f;
        //const float beta = clear ? 0.0f : 1.0f;
        //hipblasGemmEx(cublas_handle,
        //             HIPBLAS_OP_N, HIPBLAS_OP_N,
        //             size_n, size_m, size_k,
        //             &alpha, temp_dq, HIP_R_16F, size_n,
        //                     a,       HIP_R_16F, size_k,
        //             &beta,  c,       HIP_R_16F, size_n,
        //             HIP_R_16F, CUBLAS_GEMM_DFALT_TENSOR_OP);
    }
    else
    {
        // Quantized matmul

        int block_m_size_max = b->is_gptq ? GPTQ_BLOCK_M_SIZE_MAX : EXL2_BLOCK_M_SIZE_MAX;
        int max_chunks = size_m / block_m_size_max;
        int last_chunk = max_chunks * block_m_size_max;
        int last_chunk_size = size_m - last_chunk;

        if (max_chunks)
        {
            gemm_half_q_half_cuda_part(a, b, c, last_chunk, size_n, size_k, block_m_size_max, clear, r_weights, r_weights_stride, mul_r_weights);
        }

        if (last_chunk_size)
        {
            gemm_half_q_half_cuda_part(a + last_chunk * size_k, b, c + last_chunk * size_n, last_chunk_size, size_n, size_k, last_chunk_size, clear, r_weights, r_weights_stride, mul_r_weights);
        }
    }
}

__global__ void clear_kernel
(
    half* __restrict__ c,
    const int size_m,
    const int size_n
)
{
    int m = blockIdx.y;
    int n = (blockIdx.x * CLEAR_N_SIZE + threadIdx.x) * 8;
    if (n >= size_n) return;
    int4* c_ptr = (int4*)(c + m * size_n + n);
    *c_ptr = {};
}

void clear_tensor_cuda
(
    half* c,
    int size_m,
    int size_n
)
{
//     dim3 blockDim, gridDim;
//     blockDim.x = CLEAR_N_SIZE;
//     blockDim.y = 1;
//     gridDim.x = DIVIDE(size_n / 8, CLEAR_N_SIZE);
//     gridDim.y = size_m;
//     clear_kernel<<<gridDim, blockDim>>>(c, size_m, size_n);
}
